#include "hip/hip_runtime.h"
#include "hdtCudaCollision.cuh"

#include "math.h"

namespace hdt
{
    __device__
        void subtract(const cuVector3& v1, const cuVector3& v2, cuVector3& result)
    {
        result.x = v1.x - v2.x;
        result.y = v1.y - v2.y;
        result.z = v1.z - v2.z;
        result.w = v1.w - v2.w;
    }

    __device__
        void add(const cuVector3& v1, const cuVector3& v2, cuVector3& result)
    {
        result.x = v1.x + v2.x;
        result.y = v1.y + v2.y;
        result.z = v1.z + v2.z;
        result.w = v1.w + v2.w;
    }

    __device__
        void multiply(const cuVector3& v, float c, cuVector3& result)
    {
        result.x = v.x * c;
        result.y = v.y * c;
        result.z = v.z * c;
        result.w = v.w * c;
    }

    __device__
        void crossProduct(const cuVector3& v1, const cuVector3& v2, cuVector3& result)
    {
        result.x = v1.y * v2.z - v1.z * v2.y;
        result.y = v1.z * v2.x - v1.x * v2.z;
        result.z = v1.x * v2.y - v1.y * v2.x;
    }

    __device__
        void normalize(cuVector3& v)
    {
        float mag = sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
        v.x /= mag;
        v.y /= mag;
        v.z /= mag;
    }

    __global__
        void kernelPerVertexUpdate(int n, const cuPerVertexInput* __restrict__ in, cuAabb* __restrict__ out, const cuVector3* __restrict__ vertexData)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (int i = index; i < n; i += stride)
        {
            const cuVector3& v = vertexData[in[i].vertexIndex];
            float margin = v.w * in[i].margin;

            out[i].aabbMin.x = v.x - margin;
            out[i].aabbMin.y = v.y - margin;
            out[i].aabbMin.z = v.z - margin;
            out[i].aabbMax.x = v.x + margin;
            out[i].aabbMax.y = v.y + margin;
            out[i].aabbMax.z = v.z + margin;
        }
    }

    __global__
        void kernelPerTriangleUpdate(int n, const cuPerTriangleInput* __restrict__ in, cuAabb* __restrict__ out, const cuVector3* __restrict__ vertexData)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (int i = index; i < n; i += stride)
        {
            const cuVector3& v0 = vertexData[in[i].vertexIndices[0]];
            const cuVector3& v1 = vertexData[in[i].vertexIndices[1]];
            const cuVector3& v2 = vertexData[in[i].vertexIndices[2]];

            float penetration = abs(in[i].penetration);
            float margin = max((v0.w + v1.w + v2.w) * in[i].margin / 3, penetration);

            out[i].aabbMin.x = min(v0.x, min(v1.x, v2.x)) - margin;
            out[i].aabbMin.y = min(v0.y, min(v1.y, v2.y)) - margin;
            out[i].aabbMin.z = min(v0.z, min(v1.z, v2.z)) - margin;
            out[i].aabbMax.x = max(v0.x, max(v1.x, v2.x)) + margin;
            out[i].aabbMax.y = max(v0.y, max(v1.y, v2.y)) + margin;
            out[i].aabbMax.z = max(v0.z, max(v1.z, v2.z)) + margin;
        }
    }

    __device__ cuVector3& operator+=(cuVector3& v1, cuVector3& v2)
    {
        v1.x += v2.x;
        v1.y += v2.y;
        v1.z += v2.z;
        v1.w += v2.w;
        return v1;
    }

    __device__ cuVector3 calcVertexState(const cuVector3& skinPos, const cuBone& bone, float w)
    {
        cuVector3 result;
        result.x = bone.transform[0].x * skinPos.x + bone.transform[1].x * skinPos.y + bone.transform[2].x * skinPos.z + bone.transform[3].x;
        result.y = bone.transform[0].y * skinPos.x + bone.transform[1].y * skinPos.y + bone.transform[2].y * skinPos.z + bone.transform[3].y;
        result.z = bone.transform[0].z * skinPos.x + bone.transform[1].z * skinPos.y + bone.transform[2].z * skinPos.z + bone.transform[3].z;
        result.w = bone.marginMultiplier.w;
        result.x *= w;
        result.y *= w;
        result.z *= w;
        result.w *= w;
        return result;
    }

    __global__
        void kernelBodyUpdate(int n, const cuVertex* __restrict__ in, cuVector3* __restrict__ out, const cuBone* __restrict__ boneData)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (int i = index; i < n; i += stride)
        {
            out[i] = calcVertexState(in[i].position, boneData[in[i].bones[0]], in[i].weights[0]);
            for (int j = 1; j < 4; ++j)
            {
                out[i] += calcVertexState(in[i].position, boneData[in[i].bones[j]], in[i].weights[j]);
            }
        }
    }

    __global__
        void kernelVertexVertexCollision(
            int n,
            const cuCollisionSetup* __restrict__ setup,
            cuCollisionResult* output)
    {
        extern __shared__ float sdata[];

        for (int block = blockIdx.x; block < n; block += gridDim.x)
        {
            int nA = setup[block].sizeA;
            int nB = setup[block].sizeB;
            const cuPerVertexInput* __restrict__ inA = setup[block].colliderBufA;
            const cuPerVertexInput* __restrict__ inB = setup[block].colliderBufB;
            const cuVector3* __restrict__ vertexDataA = setup[block].vertexDataA;
            const cuVector3* __restrict__ vertexDataB = setup[block].vertexDataB;

            // Depth should always be negative for collisions. Initialize it to 1 to signify no collision
            // detected.
            int tid = threadIdx.x;
            cuCollisionResult temp;
            temp.depth = 1;

            // First process each block, and keep the deepest one in temp
            for (int i = tid; i < nA * nB; i += blockDim.x)
            {
                const cuPerVertexInput& inputA = inA[i % nA];
                const cuPerVertexInput& inputB = inB[i / nA];
                const cuVector3& vA = vertexDataA[inputA.vertexIndex];
                const cuVector3& vB = vertexDataB[inputB.vertexIndex];

                float rA = vA.w * inputA.margin;
                float rB = vB.w * inputA.margin; // FIXME: Suspect this ought to be inputB, but the original code was this way
                float bound2 = (rA + rB) * (rA + rB);
                cuVector3 diff;
                subtract(vA, vB, diff);
                float dist2 = diff.x * diff.x + diff.y * diff.y + diff.z * diff.z;
                float len = sqrt(dist2);
                float dist = len - (rA + rB);
                if (dist2 <= bound2 && (dist < temp.depth))
                {
                    if (len <= FLT_EPSILON)
                    {
                        diff = { 1, 0, 0, 0 };
                    }
                    else
                    {
                        normalize(diff);
                    }
                    temp.depth = dist;
                    temp.normOnB = diff;
                    multiply(diff, rA, temp.posA);
                    multiply(diff, rB, temp.posB);
                    subtract(vA, temp.posA, temp.posA);
                    add(vB, temp.posB, temp.posB);
                    temp.colliderA = static_cast<cuCollider*>(0) + i % nA;
                    temp.colliderB = static_cast<cuCollider*>(0) + i / nA;
                }
            }

            // Set the best depth in shared data
            sdata[tid] = temp.depth;

            // Now do a reduce operation so we end up with the minimum depth in the first element
            __syncthreads();
            for (int s = blockDim.x / 2; s > 0; s >>= 1)
            {
                if (tid < s && sdata[tid] > sdata[tid + s])
                {
                    sdata[tid] = sdata[tid + s];
                }
                __syncthreads();
            }

            // If our depth is equal to the minimum, set the result. Atomic exchange ensures that only one
            // thread can do this even if there are several with the same depth.
            if (sdata[0] == temp.depth && atomicExch(sdata, 2) == temp.depth)
            {
                output[block] = temp;
            }
        }
    }

    void cuCreateStream(void** ptr)
    {
        *ptr = new hipStream_t;
        hipStreamCreate(reinterpret_cast<hipStream_t*>(*ptr));
    }

    void cuDestroyStream(void* ptr)
    {
        hipStreamDestroy(*reinterpret_cast<hipStream_t*>(ptr));
        delete reinterpret_cast<hipStream_t*>(ptr);
    }

    void cuGetDeviceBuffer(void** buf, int size)
    {
        hipMalloc(buf, size);
    }

    void cuGetHostBuffer(void** buf, int size)
    {
        hipHostMalloc(buf, size);
    }

    void cuFreeDevice(void* buf)
    {
        hipFree(buf);
    }

    void cuFreeHost(void* buf)
    {
        hipHostFree(buf);
    }

    void cuCopyToDevice(void* dst, void* src, size_t n, void* stream)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        hipMemcpyAsync(dst, src, n, hipMemcpyHostToDevice, *s);
    }

    void cuCopyToHost(void* dst, void* src, size_t n, void* stream)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        hipMemcpyAsync(dst, src, n, hipMemcpyDeviceToHost, *s);
    }

    bool cuRunBodyUpdate(void* stream, int n, cuVertex* input, cuVector3* output, cuBone* boneData)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        int numBlocks = (n - 1) / cuBlockSize() + 1;

        kernelBodyUpdate <<<numBlocks, cuBlockSize(), 0, *s >>> (n, input, output, boneData);
        return hipPeekAtLastError() == hipSuccess;
    }

    bool cuRunPerVertexUpdate(void* stream, int n, cuPerVertexInput* input, cuAabb* output, cuVector3* vertexData)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        int numBlocks = (n - 1) / cuBlockSize() + 1;

        kernelPerVertexUpdate <<<numBlocks, cuBlockSize(), 0, *s >>> (n, input, output, vertexData);
        return hipPeekAtLastError() == hipSuccess;
    }


    bool cuRunPerTriangleUpdate(void* stream, int n, cuPerTriangleInput* input, cuAabb* output, cuVector3* vertexData)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        int numBlocks = (n - 1) / cuBlockSize() + 1;

        kernelPerTriangleUpdate <<<numBlocks, cuBlockSize(), 0, *s >>> (n, input, output, vertexData);
        return hipPeekAtLastError() == hipSuccess;
    }

    bool cuRunCollision(void* stream, int n, cuCollisionSetup* setup, cuCollisionResult* output)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);

        kernelVertexVertexCollision <<<n, cuBlockSize(), cuBlockSize() * sizeof(float), *s >>> (n, setup, output);
        return hipPeekAtLastError() == hipSuccess;
    }

    bool cuRunCollision(void* stream, int nA, int nB, cuPerVertexInput* inA, cuPerTriangleInput* inB, cuCollisionResult* output, cuVector3* vertexDataA, cuVector3* vertexDataB)
    {
        return true;
    }

    bool cuSynchronize(void* stream)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);

        if (s)
        {
            return hipStreamSynchronize(*s);
        }
        else
        {
            return hipDeviceSynchronize() == hipSuccess;
        }
    }

    void cuCreateEvent(void** ptr)
    {
        *ptr = new hipEvent_t;
        hipEventCreate(reinterpret_cast<hipEvent_t*>(*ptr));
    }

    void cuDestroyEvent(void* ptr)
    {
        hipEventDestroy(*reinterpret_cast<hipEvent_t*>(ptr));
        delete reinterpret_cast<hipEvent_t*>(ptr);
    }

    void cuRecordEvent(void* ptr, void* stream)
    {
        hipEvent_t* e = reinterpret_cast<hipEvent_t*>(ptr);
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        hipEventRecord(*e, *s);
    }

    void cuWaitEvent(void* ptr)
    {
        hipEvent_t* e = reinterpret_cast<hipEvent_t*>(ptr);
        hipEventSynchronize(*e);
    }

    void cuInitialize()
    {
        hipSetDeviceFlags(hipDeviceScheduleYield);
    }
}
