#include "hip/hip_runtime.h"
#include "hdtCudaCollision.cuh"

#include "math.h"

namespace hdt
{
    __device__
        void subtract(const cuVector3& v1, const cuVector3& v2, cuVector3& result)
    {
        result.x = v1.x - v2.x;
        result.y = v1.y - v2.y;
        result.z = v1.z - v2.z;
        result.w = v1.w - v2.w;
    }

    __device__
        void add(const cuVector3& v1, const cuVector3& v2, cuVector3& result)
    {
        result.x = v1.x + v2.x;
        result.y = v1.y + v2.y;
        result.z = v1.z + v2.z;
        result.w = v1.w + v2.w;
    }

    __device__
        void multiply(const cuVector3& v, float c, cuVector3& result)
    {
        result.x = v.x * c;
        result.y = v.y * c;
        result.z = v.z * c;
        result.w = v.w * c;
    }

    __device__
        void crossProduct(const cuVector3& v1, const cuVector3& v2, cuVector3& result)
    {
        result.x = v1.y * v2.z - v1.z * v2.y;
        result.y = v1.z * v2.x - v1.x * v2.z;
        result.z = v1.x * v2.y - v1.y * v2.x;
    }

    __device__
        void normalize(cuVector3& v)
    {
        float mag = sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
        v.x /= mag;
        v.y /= mag;
        v.z /= mag;
    }

    __global__
        void kernelPerVertexUpdate(int n, const cuPerVertexInput* __restrict__ in, cuAabb* __restrict__ out, const cuVector3* __restrict__ vertexData)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (int i = index; i < n; i += stride)
        {
            const cuVector3& v = vertexData[in[i].vertexIndex];
            float margin = v.w * in[i].margin;

            out[i].aabbMin.x = v.x - margin;
            out[i].aabbMin.y = v.y - margin;
            out[i].aabbMin.z = v.z - margin;
            out[i].aabbMax.x = v.x + margin;
            out[i].aabbMax.y = v.y + margin;
            out[i].aabbMax.z = v.z + margin;
        }
    }

    __global__
        void kernelPerTriangleUpdate(int n, const cuPerTriangleInput* __restrict__ in, cuAabb* __restrict__ out, const cuVector3* __restrict__ vertexData)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (int i = index; i < n; i += stride)
        {
            const cuVector3& v0 = vertexData[in[i].vertexIndices[0]];
            const cuVector3& v1 = vertexData[in[i].vertexIndices[1]];
            const cuVector3& v2 = vertexData[in[i].vertexIndices[2]];

            float penetration = abs(in[i].penetration);
            float margin = max((v0.w + v1.w + v2.w) * in[i].margin / 3, penetration);

            out[i].aabbMin.x = min(v0.x, min(v1.x, v2.x)) - margin;
            out[i].aabbMin.y = min(v0.y, min(v1.y, v2.y)) - margin;
            out[i].aabbMin.z = min(v0.z, min(v1.z, v2.z)) - margin;
            out[i].aabbMax.x = max(v0.x, max(v1.x, v2.x)) + margin;
            out[i].aabbMax.y = max(v0.y, max(v1.y, v2.y)) + margin;
            out[i].aabbMax.z = max(v0.z, max(v1.z, v2.z)) + margin;
        }
    }

    __device__ cuVector3& operator+=(cuVector3& v1, cuVector3& v2)
    {
        v1.x += v2.x;
        v1.y += v2.y;
        v1.z += v2.z;
        v1.w += v2.w;
        return v1;
    }

    __device__ cuVector3 calcVertexState(const cuVector3& skinPos, const cuBone& bone, float w)
    {
        cuVector3 result;
        result.x = bone.transform[0].x * skinPos.x + bone.transform[1].x * skinPos.y + bone.transform[2].x * skinPos.z + bone.transform[3].x;
        result.y = bone.transform[0].y * skinPos.x + bone.transform[1].y * skinPos.y + bone.transform[2].y * skinPos.z + bone.transform[3].y;
        result.z = bone.transform[0].z * skinPos.x + bone.transform[1].z * skinPos.y + bone.transform[2].z * skinPos.z + bone.transform[3].z;
        result.w = bone.marginMultiplier.w;
        result.x *= w;
        result.y *= w;
        result.z *= w;
        result.w *= w;
        return result;
    }

    __global__
        void kernelBodyUpdate(int n, const cuVertex* __restrict__ in, cuVector3* __restrict__ out, const cuBone* __restrict__ boneData)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (int i = index; i < n; i += stride)
        {
            out[i] = calcVertexState(in[i].position, boneData[in[i].bones[0]], in[i].weights[0]);
            for (int j = 1; j < 4; ++j)
            {
                out[i] += calcVertexState(in[i].position, boneData[in[i].bones[j]], in[i].weights[j]);
            }
        }
    }

    // collidePair does the actual collision between two colliders, always a vertex and some other type. It
    // should modify output if and only if there is a collision.
    __device__ bool collidePair(
        const cuPerVertexInput& __restrict__ inputA,
        const cuPerVertexInput& __restrict__ inputB,
        const cuVector3* __restrict__ vertexDataA,
        const cuVector3* __restrict__ vertexDataB,
        cuCollisionResult& output)
    {
        const cuVector3& vA = vertexDataA[inputA.vertexIndex];
        const cuVector3& vB = vertexDataB[inputB.vertexIndex];

        float rA = vA.w * inputA.margin;
        float rB = vB.w * inputB.margin;
        float bound2 = (rA + rB) * (rA + rB);
        cuVector3 diff;
        subtract(vA, vB, diff);
        float dist2 = diff.x * diff.x + diff.y * diff.y + diff.z * diff.z;
        float len = sqrt(dist2);
        float dist = len - (rA + rB);
        if (dist2 <= bound2 && (dist < output.depth))
        {
            if (len <= FLT_EPSILON)
            {
                diff = { 1, 0, 0, 0 };
            }
            else
            {
                normalize(diff);
            }
            output.depth = dist;
            output.normOnB = diff;
            multiply(diff, rA, output.posA);
            multiply(diff, rB, output.posB);
            subtract(vA, output.posA, output.posA);
            add(vB, output.posB, output.posB);
            return true;
        }
        return false;
    }

    __device__ bool collidePair(
        const cuPerVertexInput& __restrict__ inputA,
        const cuPerTriangleInput& __restrict__ inputB,
        const cuVector3* __restrict__ vertexDataA,
        const cuVector3* __restrict__ vertexDataB,
        cuCollisionResult& output)
    {
        return false;
    }

    // kernelCollision does the supporting work for threading the collision checks and making sure that only
    // the deepest result is kept.
    template <typename T>
    __global__ void kernelCollision(
        int n,
        const cuCollisionSetup<T>* __restrict__ setup,
        cuCollisionResult* output)
    {
        extern __shared__ float sdata[];

        for (int block = blockIdx.x; block < n; block += gridDim.x)
        {
            int nA = setup[block].sizeA;
            int nB = setup[block].sizeB;
            const cuPerVertexInput* __restrict__ inA = setup[block].colliderBufA;
            const auto* __restrict__ inB = setup[block].colliderBufB;
            const cuVector3* __restrict__ vertexDataA = setup[block].vertexDataA;
            const cuVector3* __restrict__ vertexDataB = setup[block].vertexDataB;

            // Depth should always be negative for collisions. We'll use positive values to signify no
            // collision, and later for mutual exclusion.
            int tid = threadIdx.x;
            cuCollisionResult temp;
            temp.depth = 1;

            for (int i = tid; i < nA * nB; i += blockDim.x)
            {
                if (collidePair(inA[i % nA], inB[i / nA], vertexDataA, vertexDataB, temp))
                {
                    temp.colliderA = static_cast<cuCollider*>(0) + i % nA;
                    temp.colliderB = static_cast<cuCollider*>(0) + i / nA;
                }
            }

            // Set the best depth for this thread in shared memory
            sdata[tid] = temp.depth;

            // Now reduce to find the minimum depth, and store it in the first element
            __syncthreads();
            for (int s = blockDim.x / 2; s > 0; s >>= 1)
            {
                if (tid < s && sdata[tid] > sdata[tid + s])
                {
                    sdata[tid] = sdata[tid + s];
                }
                __syncthreads();
            }

            // If the depth of this thread is equal to the minimum, try to set the result. Do an atomic
            // exchange with the first value to ensure that only one thread gets to do this in case of ties.
            if (sdata[0] == temp.depth && atomicExch(sdata, 2) == temp.depth)
            {
                output[block] = temp;
            }
        }
    }

    void cuCreateStream(void** ptr)
    {
        *ptr = new hipStream_t;
        hipStreamCreate(reinterpret_cast<hipStream_t*>(*ptr));
    }

    void cuDestroyStream(void* ptr)
    {
        hipStreamDestroy(*reinterpret_cast<hipStream_t*>(ptr));
        delete reinterpret_cast<hipStream_t*>(ptr);
    }

    void cuGetDeviceBuffer(void** buf, int size)
    {
        hipMalloc(buf, size);
    }

    void cuGetHostBuffer(void** buf, int size)
    {
        hipHostMalloc(buf, size);
    }

    void cuFreeDevice(void* buf)
    {
        hipFree(buf);
    }

    void cuFreeHost(void* buf)
    {
        hipHostFree(buf);
    }

    void cuCopyToDevice(void* dst, void* src, size_t n, void* stream)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        hipMemcpyAsync(dst, src, n, hipMemcpyHostToDevice, *s);
    }

    void cuCopyToHost(void* dst, void* src, size_t n, void* stream)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        hipMemcpyAsync(dst, src, n, hipMemcpyDeviceToHost, *s);
    }

    bool cuRunBodyUpdate(void* stream, int n, cuVertex* input, cuVector3* output, cuBone* boneData)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        int numBlocks = (n - 1) / cuBlockSize() + 1;

        kernelBodyUpdate <<<numBlocks, cuBlockSize(), 0, *s >>> (n, input, output, boneData);
        return hipPeekAtLastError() == hipSuccess;
    }

    bool cuRunPerVertexUpdate(void* stream, int n, cuPerVertexInput* input, cuAabb* output, cuVector3* vertexData)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        int numBlocks = (n - 1) / cuBlockSize() + 1;

        kernelPerVertexUpdate <<<numBlocks, cuBlockSize(), 0, *s >>> (n, input, output, vertexData);
        return hipPeekAtLastError() == hipSuccess;
    }


    bool cuRunPerTriangleUpdate(void* stream, int n, cuPerTriangleInput* input, cuAabb* output, cuVector3* vertexData)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        int numBlocks = (n - 1) / cuBlockSize() + 1;

        kernelPerTriangleUpdate <<<numBlocks, cuBlockSize(), 0, *s >>> (n, input, output, vertexData);
        return hipPeekAtLastError() == hipSuccess;
    }

    template<typename T>
    bool cuRunCollision(void* stream, int n, cuCollisionSetup<T>* setup, cuCollisionResult* output)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);

        kernelCollision <<<n, cuBlockSize(), cuBlockSize() * sizeof(float), *s >>> (n, setup, output);
        return hipPeekAtLastError() == hipSuccess;
    }

    bool cuSynchronize(void* stream)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);

        if (s)
        {
            return hipStreamSynchronize(*s);
        }
        else
        {
            return hipDeviceSynchronize() == hipSuccess;
        }
    }

    void cuCreateEvent(void** ptr)
    {
        *ptr = new hipEvent_t;
        hipEventCreate(reinterpret_cast<hipEvent_t*>(*ptr));
    }

    void cuDestroyEvent(void* ptr)
    {
        hipEventDestroy(*reinterpret_cast<hipEvent_t*>(ptr));
        delete reinterpret_cast<hipEvent_t*>(ptr);
    }

    void cuRecordEvent(void* ptr, void* stream)
    {
        hipEvent_t* e = reinterpret_cast<hipEvent_t*>(ptr);
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        hipEventRecord(*e, *s);
    }

    void cuWaitEvent(void* ptr)
    {
        hipEvent_t* e = reinterpret_cast<hipEvent_t*>(ptr);
        hipEventSynchronize(*e);
    }

    void cuInitialize()
    {
        hipSetDeviceFlags(hipDeviceScheduleYield);
    }

    template bool cuRunCollision<CudaPerVertexShape>(void*, int, cuCollisionSetup<CudaPerVertexShape>*, cuCollisionResult*);
    template bool cuRunCollision<CudaPerTriangleShape>(void*, int, cuCollisionSetup<CudaPerTriangleShape>*, cuCollisionResult*);
}
