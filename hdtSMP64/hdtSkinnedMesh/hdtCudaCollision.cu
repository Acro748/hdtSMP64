#include "hip/hip_runtime.h"
#include "hdtCudaCollision.cuh"

#include "math.h"

// Check collider bounding boxes on the GPU. This reduces the total amount of work, but is bad for
// divergence and increases register usage.
#define GPU_BOUNDING_BOX_CHECK

namespace hdt
{
    constexpr int cuBlockSize() { return 1024; }

    // Reduction makes heavy use of shared memory (16 bytes per thread), so a smaller block size may be desirable
    constexpr int cuReduceBlockSize() { return 1024; }

    template<typename T>
    constexpr int collisionBlockSize();

    // Collision checking is quite register-hungry, so we may need to reduce the block size for it here
    template<>
    constexpr int collisionBlockSize<cuPerVertexInput>() { return 1024; }
    template<>
    constexpr int collisionBlockSize<cuPerTriangleInput>() { return 1024; }

    __device__
        void subtract(const cuVector3& v1, const cuVector3& v2, cuVector3& result)
    {
        result.x = v1.x - v2.x;
        result.y = v1.y - v2.y;
        result.z = v1.z - v2.z;
        result.w = v1.w - v2.w;
    }

    __device__
        void add(const cuVector3& v1, const cuVector3& v2, cuVector3& result)
    {
        result.x = v1.x + v2.x;
        result.y = v1.y + v2.y;
        result.z = v1.z + v2.z;
        result.w = v1.w + v2.w;
    }

    __device__
        void multiply(const cuVector3& v, float c, cuVector3& result)
    {
        result.x = v.x * c;
        result.y = v.y * c;
        result.z = v.z * c;
        result.w = v.w * c;
    }

    __device__
        void crossProduct(const cuVector3& v1, const cuVector3& v2, cuVector3& result)
    {
        result.x = v1.y * v2.z - v1.z * v2.y;
        result.y = v1.z * v2.x - v1.x * v2.z;
        result.z = v1.x * v2.y - v1.y * v2.x;
    }

    __device__
        float dotProduct(const cuVector3& v1, const cuVector3& v2)
    {
        return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
    }

    __device__
        float magnitude(const cuVector3& v)
    {
        return sqrt(dotProduct(v, v));
    }

    __device__
        void normalize(cuVector3& v)
    {
        float mag = magnitude(v);
        v.x /= mag;
        v.y /= mag;
        v.z /= mag;
    }

    __device__
        bool boundingBoxCollision(const cuAabb& b1, const cuAabb& b2)
    {
        return !(b1.aabbMin.x > b2.aabbMax.x ||
            b1.aabbMin.y > b2.aabbMax.y ||
            b1.aabbMin.z > b2.aabbMax.z ||
            b1.aabbMax.x < b2.aabbMin.x ||
            b1.aabbMax.y < b2.aabbMin.y ||
            b1.aabbMax.z < b2.aabbMin.z);
    }

    __global__
        void kernelPerVertexUpdate(int n, const cuPerVertexInput* __restrict__ in, cuAabb* __restrict__ out, const cuVector3* __restrict__ vertexData)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (int i = index; i < n; i += stride)
        {
            const cuVector3& v = vertexData[in[i].vertexIndex];
            float margin = v.w * in[i].margin;

            out[i].aabbMin.x = v.x - margin;
            out[i].aabbMin.y = v.y - margin;
            out[i].aabbMin.z = v.z - margin;
            out[i].aabbMax.x = v.x + margin;
            out[i].aabbMax.y = v.y + margin;
            out[i].aabbMax.z = v.z + margin;
        }
    }

    __global__
        void kernelPerTriangleUpdate(int n, const cuPerTriangleInput* __restrict__ in, cuAabb* __restrict__ out, const cuVector3* __restrict__ vertexData)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (int i = index; i < n; i += stride)
        {
            const cuVector3& v0 = vertexData[in[i].vertexIndices[0]];
            const cuVector3& v1 = vertexData[in[i].vertexIndices[1]];
            const cuVector3& v2 = vertexData[in[i].vertexIndices[2]];

            float penetration = abs(in[i].penetration);
            float margin = max((v0.w + v1.w + v2.w) * in[i].margin / 3, penetration);

            out[i].aabbMin.x = min(v0.x, min(v1.x, v2.x)) - margin;
            out[i].aabbMin.y = min(v0.y, min(v1.y, v2.y)) - margin;
            out[i].aabbMin.z = min(v0.z, min(v1.z, v2.z)) - margin;
            out[i].aabbMax.x = max(v0.x, max(v1.x, v2.x)) + margin;
            out[i].aabbMax.y = max(v0.y, max(v1.y, v2.y)) + margin;
            out[i].aabbMax.z = max(v0.z, max(v1.z, v2.z)) + margin;
        }
    }

    __global__
        void kernelBoundingBoxReduce(int n, const std::pair<int, int>* __restrict__ nodeData, const cuAabb* __restrict__ boundingBoxes, cuAabb* output)
    {
        extern __shared__ cuVector3 shared[];

        for (int block = blockIdx.x; block < n; block += gridDim.x)
        {
            const cuAabb* aabbStart = boundingBoxes + nodeData[block].first;
            int aabbCount = nodeData[block].second;
            int tid = threadIdx.x;

            // To reduce demand on shared memory, process min and max separately. We may also need to process
            // the data in multiple blocks - we use the fast divide-and-conquer approach within a block, but
            // combine them linearly.
            cuVector3 temp = { FLT_MAX, FLT_MAX, FLT_MAX, 0 };
            for (int i = tid; i < aabbCount; i += 2 * blockDim.x)
            {
                // First step takes data from the individual bounding boxes and populates shared memory
                int s = blockDim.x;
                if (i + s < aabbCount)
                {
                    shared[tid].x = min(aabbStart[i].aabbMin.x, aabbStart[i + s].aabbMin.x);
                    shared[tid].y = min(aabbStart[i].aabbMin.y, aabbStart[i + s].aabbMin.y);
                    shared[tid].z = min(aabbStart[i].aabbMin.z, aabbStart[i + s].aabbMin.z);
                }
                else
                {
                    shared[tid] = aabbStart[i].aabbMin;
                }

                // Now we can do a conventional reduction
                s >>= 1;
                __syncthreads();
                for (; s > 0; s >>= 1)
                {
                    if (tid < s && i + s < aabbCount)
                    {
                        shared[tid].x = min(shared[tid].x, shared[tid + s].x);
                        shared[tid].y = min(shared[tid].y, shared[tid + s].y);
                        shared[tid].z = min(shared[tid].z, shared[tid + s].z);
                    }
                    __syncthreads();
                }

                // Finally, thread 0 combines with the result from previous blocks
                if (tid == 0)
                {
                    temp.x = min(temp.x, shared[tid].x);
                    temp.y = min(temp.x, shared[tid].y);
                    temp.z = min(temp.x, shared[tid].z);
                }
            }
            if (tid == 0)
            {
                output[block].aabbMin = temp;
            }

            // Now do the same again for the maximums
            temp = { -FLT_MAX, -FLT_MAX, -FLT_MAX, 0 };
            for (int i = tid; i < aabbCount; i += 2 * blockDim.x)
            {
                int s = blockDim.x;
                if (i + s < aabbCount)
                {
                    shared[tid].x = max(aabbStart[i].aabbMax.x, aabbStart[i + s].aabbMax.x);
                    shared[tid].y = max(aabbStart[i].aabbMax.y, aabbStart[i + s].aabbMax.y);
                    shared[tid].z = max(aabbStart[i].aabbMax.z, aabbStart[i + s].aabbMax.z);
                }
                else
                {
                    shared[tid] = aabbStart[i].aabbMin;
                }
                s >>= 1;
                __syncthreads();
                for (; s > 0; s >>= 1)
                {
                    if (tid < s && i + s < aabbCount)
                    {
                        shared[tid].x = max(shared[tid].x, shared[tid + s].x);
                        shared[tid].y = max(shared[tid].y, shared[tid + s].y);
                        shared[tid].z = max(shared[tid].z, shared[tid + s].z);
                    }
                    __syncthreads();
                }
                if (tid == 0)
                {
                    temp.x = max(temp.x, shared[tid].x);
                    temp.y = max(temp.x, shared[tid].y);
                    temp.z = max(temp.x, shared[tid].z);
                }
            }
            if (tid == 0)
            {
                output[block].aabbMax = temp;
            }
        }
    }

    __device__ cuVector3& operator+=(cuVector3& v1, cuVector3& v2)
    {
        v1.x += v2.x;
        v1.y += v2.y;
        v1.z += v2.z;
        v1.w += v2.w;
        return v1;
    }

    __device__ cuVector3 calcVertexState(const cuVector3& skinPos, const cuBone& bone, float w)
    {
        cuVector3 result;
        result.x = bone.transform[0].x * skinPos.x + bone.transform[1].x * skinPos.y + bone.transform[2].x * skinPos.z + bone.transform[3].x;
        result.y = bone.transform[0].y * skinPos.x + bone.transform[1].y * skinPos.y + bone.transform[2].y * skinPos.z + bone.transform[3].y;
        result.z = bone.transform[0].z * skinPos.x + bone.transform[1].z * skinPos.y + bone.transform[2].z * skinPos.z + bone.transform[3].z;
        result.w = bone.marginMultiplier.w;
        result.x *= w;
        result.y *= w;
        result.z *= w;
        result.w *= w;
        return result;
    }

    __global__
        void kernelBodyUpdate(int n, const cuVertex* __restrict__ in, cuVector3* __restrict__ out, const cuBone* __restrict__ boneData)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (int i = index; i < n; i += stride)
        {
            out[i] = calcVertexState(in[i].position, boneData[in[i].bones[0]], in[i].weights[0]);
            for (int j = 1; j < 4; ++j)
            {
                out[i] += calcVertexState(in[i].position, boneData[in[i].bones[j]], in[i].weights[j]);
            }
        }
    }

    // collidePair does the actual collision between two colliders, always a vertex and some other type. It
    // should modify output if and only if there is a collision.
    __device__ bool collidePair(
        const cuPerVertexInput& __restrict__ inputA,
        const cuPerVertexInput& __restrict__ inputB,
        const cuVector3* __restrict__ vertexDataA,
        const cuVector3* __restrict__ vertexDataB,
        cuCollisionResult& output)
    {
        const cuVector3& vA = vertexDataA[inputA.vertexIndex];
        const cuVector3& vB = vertexDataB[inputB.vertexIndex];

        float rA = vA.w * inputA.margin;
        float rB = vB.w * inputB.margin;
        float bound2 = (rA + rB) * (rA + rB);
        cuVector3 diff;
        subtract(vA, vB, diff);
        float dist2 = dotProduct(diff, diff);
        float len = sqrt(dist2);
        float dist = len - (rA + rB);
        if (dist2 <= bound2 && (dist < output.depth))
        {
            if (len <= FLT_EPSILON)
            {
                diff = { 1, 0, 0, 0 };
            }
            else
            {
                normalize(diff);
            }
            output.depth = dist;
            output.normOnB = diff;
            multiply(diff, rA, output.posA);
            multiply(diff, rB, output.posB);
            subtract(vA, output.posA, output.posA);
            add(vB, output.posB, output.posB);
            return true;
        }
        return false;
    }

    __device__ bool collidePair(
        const cuPerVertexInput& __restrict__ inputA,
        const cuPerTriangleInput& __restrict__ inputB,
        const cuVector3* __restrict__ vertexDataA,
        const cuVector3* __restrict__ vertexDataB,
        cuCollisionResult& output)
    {
        cuVector3 s = vertexDataA[inputA.vertexIndex];
        float r = s.w * inputA.margin;
        cuVector3 p0 = vertexDataB[inputB.vertexIndices[0]];
        cuVector3 p1 = vertexDataB[inputB.vertexIndices[1]];
        cuVector3 p2 = vertexDataB[inputB.vertexIndices[2]];
        float margin = (p0.w + p1.w + p2.w) / 3.0;
        float penetration = inputB.penetration * margin;
        margin *= inputB.margin;
        if (penetration > -FLT_EPSILON && penetration < FLT_EPSILON)
        {
            penetration = 0;
        }

        // Compute unit normal and twice area of triangle
        cuVector3 ab;
        cuVector3 ac;
        subtract(p1, p0, ab);
        subtract(p2, p0, ac);
        cuVector3 normal;
        crossProduct(ab, ac, normal);
        float area = magnitude(normal);
        if (area < FLT_EPSILON)
        {
            return false;
        }
        multiply(normal, 1.0 / area, normal);

        // Reverse normal direction if penetration is negative
        if (penetration < 0)
        {
            multiply(normal, -1.0, normal);
            penetration = -penetration;
        }

        // Compute distance from point to plane and its projection onto the plane
        cuVector3 ap;
        subtract(s, p0, ap);
        float distance = dotProduct(ap, normal);
        cuVector3 projection;
        multiply(normal, distance, projection);
        subtract(s, projection, projection);

        // Determine whether the point is close enough to the plane
        float radiusWithMargin = r + margin;
        if (penetration >= FLT_EPSILON)
        {
            if (distance >= radiusWithMargin || distance < -penetration)
            {
                return false;
            }
        }
        else
        {
            if (distance < 0)
            {
                distance = -distance;
                multiply(normal, -1, normal);
            }
            if (distance >= radiusWithMargin)
            {
                return false;
            }
        }

        // Don't bother to do any more if depth isn't negative, or we already have a deeper collision
        float depth = distance - radiusWithMargin;
        if (depth >= -FLT_EPSILON || depth >= output.depth)
        {
            return false;
        }

        // Compute twice the area of each triangle formed by the projection
        cuVector3 bp;
        cuVector3 cp;
        subtract(projection, p0, ap);
        subtract(projection, p1, bp);
        subtract(projection, p2, cp);
        cuVector3 aa;
        crossProduct(bp, cp, aa);
        crossProduct(cp, ap, ab);
        crossProduct(ap, bp, ac);
        float areaA = magnitude(aa);
        float areaB = magnitude(ab);
        float areaC = magnitude(ac);
        if (areaA + areaB > area || areaB + areaC > area || areaC + areaA > area)
        {
            return false;
        }

        // FIXME: posA doesn't take the margin into account here
        output.normOnB = normal;
        output.posB = projection;
        multiply(normal, r, projection);
        subtract(s, projection, output.posA);
        output.depth = depth;
        return true;
    }

    // kernelCollision does the supporting work for threading the collision checks and making sure that only
    // the deepest result is kept.
    template <typename T>
    __global__ void kernelCollision(
        int n,
        const cuCollisionSetup* __restrict__ setup,
        const cuPerVertexInput* __restrict__ inA,
        const T* __restrict__ inB,
        const cuAabb* __restrict__ boundingBoxesA,
        const cuAabb* __restrict__ boundingBoxesB,
        const cuVector3* __restrict__ vertexDataA,
        const cuVector3* __restrict__ vertexDataB,
        cuCollisionResult* output)
    {
        extern __shared__ float sdata[];

        for (int block = blockIdx.x; block < n; block += gridDim.x)
        {
            int nA = setup[block].sizeA;
            int nB = setup[block].sizeB;
            int* indicesA = setup[block].indicesA;
            int* indicesB = setup[block].indicesB;

            // Depth should always be negative for collisions. We'll use positive values to signify no
            // collision, and later for mutual exclusion.
            int tid = threadIdx.x;
            cuCollisionResult temp;
            temp.depth = 1;

            int nPairs = nA * nB;
            for (int i = tid; i < nPairs; i += blockDim.x)
            {
                int iA = indicesA[i % nA];
                int iB = indicesB[i / nA];

                // Skip pairs until we find one with a bounding box collision. This should increase the
                // number of full checks done in parallel, and reduce divergence overall. Note we only do
                // this at all if there are more pairs than threads - if there's only enough work for a
                // single iteration (very common), there's no benefit to trying to reduce it.
#ifdef GPU_BOUNDING_BOX_CHECK
                if (nPairs > blockDim.x)
                {
                    while (i < nPairs && !boundingBoxCollision(boundingBoxesA[iA], boundingBoxesB[iB]))
                    {
                        i += blockDim.x;
                        iA = indicesA[i % nA];
                        iB = indicesB[i / nA];
                    }
                }
#endif

                if (i < nPairs && collidePair(inA[iA], inB[iB], vertexDataA, vertexDataB, temp))
                {
                    temp.colliderA = static_cast<cuCollider*>(0) + iA;
                    temp.colliderB = static_cast<cuCollider*>(0) + iB;
                }
            }

            // Set the best depth for this thread in shared memory
            sdata[tid] = temp.depth;

            // Now reduce to find the minimum depth, and store it in the first element
            __syncthreads();
            for (int s = blockDim.x / 2; s > 0; s >>= 1)
            {
                if (tid < s && sdata[tid] > sdata[tid + s])
                {
                    sdata[tid] = sdata[tid + s];
                }
                __syncthreads();
            }

            // If the depth of this thread is equal to the minimum, try to set the result. Do an atomic
            // exchange with the first value to ensure that only one thread gets to do this in case of ties.
            if (sdata[0] == temp.depth && atomicExch(sdata, 2) == temp.depth)
            {
                output[block] = temp;
            }
        }
    }

    void cuCreateStream(void** ptr)
    {
        *ptr = new hipStream_t;
        hipStreamCreate(reinterpret_cast<hipStream_t*>(*ptr));
    }

    void cuDestroyStream(void* ptr)
    {
        hipStreamDestroy(*reinterpret_cast<hipStream_t*>(ptr));
        delete reinterpret_cast<hipStream_t*>(ptr);
    }

    void cuGetDeviceBuffer(void** buf, int size)
    {
        hipMalloc(buf, size);
    }

    void cuGetHostBuffer(void** buf, int size)
    {
        hipHostMalloc(buf, size);
    }

    void cuFreeDevice(void* buf)
    {
        hipFree(buf);
    }

    void cuFreeHost(void* buf)
    {
        hipHostFree(buf);
    }

    void cuCopyToDevice(void* dst, void* src, size_t n, void* stream)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        hipMemcpyAsync(dst, src, n, hipMemcpyHostToDevice, *s);
    }

    void cuCopyToHost(void* dst, void* src, size_t n, void* stream)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        hipMemcpyAsync(dst, src, n, hipMemcpyDeviceToHost, *s);
    }

    bool cuRunBodyUpdate(void* stream, int n, cuVertex* input, cuVector3* output, cuBone* boneData)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        int numBlocks = (n - 1) / cuBlockSize() + 1;

        kernelBodyUpdate <<<numBlocks, cuBlockSize(), 0, *s >>> (n, input, output, boneData);
        return hipPeekAtLastError() == hipSuccess;
    }

    bool cuRunPerVertexUpdate(void* stream, int n, cuPerVertexInput* input, cuAabb* output, cuVector3* vertexData)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        int numBlocks = (n - 1) / cuBlockSize() + 1;

        kernelPerVertexUpdate <<<numBlocks, cuBlockSize(), 0, *s >>> (n, input, output, vertexData);
        return hipPeekAtLastError() == hipSuccess;
    }


    bool cuRunPerTriangleUpdate(void* stream, int n, cuPerTriangleInput* input, cuAabb* output, cuVector3* vertexData)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        int numBlocks = (n - 1) / cuBlockSize() + 1;

        kernelPerTriangleUpdate <<<numBlocks, cuBlockSize(), 0, *s >>> (n, input, output, vertexData);
        return hipPeekAtLastError() == hipSuccess;
    }

    template<typename T>
    bool cuRunCollision(
        void* stream,
        int n,
        cuCollisionSetup* setup,
        cuPerVertexInput* inA,
        T* inB,
        cuAabb* boundingBoxesA,
        cuAabb* boundingBoxesB,
        cuVector3* vertexDataA,
        cuVector3* vertexDataB,
        cuCollisionResult* output)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);

        kernelCollision <<<n, collisionBlockSize<T>(), collisionBlockSize<T>() * sizeof(float), *s >>> (
            n, setup, inA, inB, boundingBoxesA, boundingBoxesB, vertexDataA, vertexDataB, output);
        return hipPeekAtLastError() == hipSuccess;
    }

    bool cuRunBoundingBoxReduce(void* stream, int n, int largestNode, std::pair<int, int>* setup, cuAabb* boundingBoxes, cuAabb* output)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);

        // Block size for bounding box reduction should be half the size of the largest block, rounded up to
        // a multiple of 32, not exceeding the maximum block size.
        int blockSize = min(32 * ((largestNode - 1) / 64 + 1), cuReduceBlockSize());
        
        kernelBoundingBoxReduce <<<n, blockSize, blockSize * sizeof(cuVector3), *s >>> (n, setup, boundingBoxes, output);
        return hipPeekAtLastError() == hipSuccess;
    }

    bool cuSynchronize(void* stream)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);

        if (s)
        {
            return hipStreamSynchronize(*s);
        }
        else
        {
            return hipDeviceSynchronize() == hipSuccess;
        }
    }

    void cuCreateEvent(void** ptr)
    {
        *ptr = new hipEvent_t;
        hipEventCreate(reinterpret_cast<hipEvent_t*>(*ptr));
    }

    void cuDestroyEvent(void* ptr)
    {
        hipEventDestroy(*reinterpret_cast<hipEvent_t*>(ptr));
        delete reinterpret_cast<hipEvent_t*>(ptr);
    }

    void cuRecordEvent(void* ptr, void* stream)
    {
        hipEvent_t* e = reinterpret_cast<hipEvent_t*>(ptr);
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        hipEventRecord(*e, *s);
    }

    void cuWaitEvent(void* ptr)
    {
        hipEvent_t* e = reinterpret_cast<hipEvent_t*>(ptr);
        hipEventSynchronize(*e);
    }

    void cuInitialize()
    {
        hipSetDeviceFlags(hipDeviceScheduleYield);
    }

    int cuDeviceCount()
    {
        int count;
        hipGetDeviceCount(&count);
        return count;
    }

    template bool cuRunCollision<cuPerVertexInput>(void*, int, cuCollisionSetup*, cuPerVertexInput*, cuPerVertexInput*, cuAabb*, cuAabb*, cuVector3*, cuVector3*, cuCollisionResult*);
    template bool cuRunCollision<cuPerTriangleInput>(void*, int, cuCollisionSetup*, cuPerVertexInput*, cuPerTriangleInput*, cuAabb*, cuAabb*, cuVector3*, cuVector3*, cuCollisionResult*);
}
