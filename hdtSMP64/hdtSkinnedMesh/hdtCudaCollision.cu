#include "hip/hip_runtime.h"
#include "hdtCudaCollision.cuh"

#include "math.h"

namespace hdt
{
    __device__
        void subtract(const cuVector3& v1, const cuVector3& v2, cuVector3& result)
    {
        result.x = v1.x - v2.x;
        result.y = v1.y - v2.y;
        result.z = v1.z - v2.z;
        result.w = v1.w - v2.w;
    }

    __device__
        void crossProduct(const cuVector3& v1, const cuVector3& v2, cuVector3& result)
    {
        result.x = v1.y * v2.z - v1.z * v2.y;
        result.y = v1.z * v2.x - v1.x * v2.z;
        result.z = v1.x * v2.y - v1.y * v2.x;
    }

    __device__
        void normalize(cuVector3& v)
    {
        float mag = sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
        v.x /= mag;
        v.y /= mag;
        v.z /= mag;
    }

    __global__
        void kernelPerVertexUpdate(int n, const cuPerVertexInput* __restrict__ in, cuAabb* __restrict__ out, const cuVector3* __restrict__ vertexData)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (int i = index; i < n; i += stride)
        {
            const cuVector3& v = vertexData[in[i].vertexIndex];
            float margin = v.w * in[i].margin;

            out[i].aabbMin.x = v.x - margin;
            out[i].aabbMin.y = v.y - margin;
            out[i].aabbMin.z = v.z - margin;
            out[i].aabbMax.x = v.x + margin;
            out[i].aabbMax.y = v.y + margin;
            out[i].aabbMax.z = v.z + margin;
        }
    }

    __global__
        void kernelPerTriangleUpdate(int n, const cuPerTriangleInput* __restrict__ in, cuAabb* __restrict__ out, const cuVector3* __restrict__ vertexData)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (int i = index; i < n; i += stride)
        {
            const cuVector3& v0 = vertexData[in[i].vertexIndices[0]];
            const cuVector3& v1 = vertexData[in[i].vertexIndices[1]];
            const cuVector3& v2 = vertexData[in[i].vertexIndices[2]];

            float penetration = abs(in[i].penetration);
            float margin = max((v0.w + v1.w + v2.w) * in[i].margin / 3, penetration);

            out[i].aabbMin.x = min(v0.x, min(v1.x, v2.x)) - margin;
            out[i].aabbMin.y = min(v0.y, min(v1.y, v2.y)) - margin;
            out[i].aabbMin.z = min(v0.z, min(v1.z, v2.z)) - margin;
            out[i].aabbMax.x = max(v0.x, max(v1.x, v2.x)) + margin;
            out[i].aabbMax.y = max(v0.y, max(v1.y, v2.y)) + margin;
            out[i].aabbMax.z = max(v0.z, max(v1.z, v2.z)) + margin;
        }
    }

    __device__ cuVector3& operator+=(cuVector3& v1, cuVector3& v2)
    {
        v1.x += v2.x;
        v1.y += v2.y;
        v1.z += v2.z;
        v1.w += v2.w;
        return v1;
    }

    __device__ cuVector3 calcVertexState(const cuVector3& skinPos, const cuBone& bone, float w)
    {
        cuVector3 result;
        result.x = bone.transform[0].x * skinPos.x + bone.transform[1].x * skinPos.y + bone.transform[2].x * skinPos.z + bone.transform[3].x;
        result.y = bone.transform[0].y * skinPos.x + bone.transform[1].y * skinPos.y + bone.transform[2].y * skinPos.z + bone.transform[3].y;
        result.z = bone.transform[0].z * skinPos.x + bone.transform[1].z * skinPos.y + bone.transform[2].z * skinPos.z + bone.transform[3].z;
        result.w = bone.marginMultiplier.w;
        result.x *= w;
        result.y *= w;
        result.z *= w;
        result.w *= w;
        return result;
    }

    __global__
        void kernelBodyUpdate(int n, const cuVertex* __restrict__ in, cuVector3* __restrict__ out, const cuBone* __restrict__ boneData)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (int i = index; i < n; i += stride)
        {
            out[i] = calcVertexState(in[i].position, boneData[in[i].bones[0]], in[i].weights[0]);
            for (int j = 1; j < 4; ++j)
            {
                out[i] += calcVertexState(in[i].position, boneData[in[i].bones[j]], in[i].weights[j]);
            }
        }
    }

    void cuCreateStream(void** ptr)
    {
        *ptr = new hipStream_t;
        hipStreamCreate(reinterpret_cast<hipStream_t*>(*ptr));
    }

    void cuDestroyStream(void* ptr)
    {
        hipStreamDestroy(*reinterpret_cast<hipStream_t*>(ptr));
        delete reinterpret_cast<hipStream_t*>(ptr);
    }

    void cuGetDeviceBuffer(void** buf, int size)
    {
        hipMalloc(buf, size);
    }

    void cuGetHostBuffer(void** buf, int size)
    {
        hipHostMalloc(buf, size);
    }

    void cuFreeDevice(void* buf)
    {
        hipFree(buf);
    }

    void cuFreeHost(void* buf)
    {
        hipHostFree(buf);
    }

    void cuCopyToDevice(void* dst, void* src, size_t n)
    {
        hipMemcpy(dst, src, n, hipMemcpyHostToDevice);
    }

    void cuCopyToHost(void* dst, void* src, size_t n)
    {
        hipMemcpy(dst, src, n, hipMemcpyDeviceToHost);
    }

    bool cuRunBodyUpdate(void* stream, int n, cuVertex* input, cuVector3* output, cuBone* boneData)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        int numBlocks = (n - 1) / 512 + 1;

        kernelBodyUpdate <<<numBlocks, 512, 0, *s >>> (n, input, output, boneData);
        return hipPeekAtLastError() == hipSuccess;
    }

    bool cuRunPerVertexUpdate(void* stream, int n, cuPerVertexInput* input, cuAabb* output, cuVector3* vertexData)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        int numBlocks = (n - 1) / 512 + 1;

        kernelPerVertexUpdate <<<numBlocks, 512, 0, *s >>> (n, input, output, vertexData);
        return hipPeekAtLastError() == hipSuccess;
    }


    bool cuRunPerTriangleUpdate(void* stream, int n, cuPerTriangleInput* input, cuAabb* output, cuVector3* vertexData)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        int numBlocks = (n - 1) / 512 + 1;

        kernelPerTriangleUpdate <<<numBlocks, 512, 0, *s >>> (n, input, output, vertexData);
        return hipPeekAtLastError() == hipSuccess;
    }

    bool cuSynchronize()
    {
        return hipDeviceSynchronize() == hipSuccess;
    }
}
