#include "hip/hip_runtime.h"
#include "hdtCudaCollision.cuh"

#include "math.h"

// Check collider bounding boxes on the GPU. This reduces the total amount of work, but is bad for
// divergence and increases register usage. Probably no longer very useful with vertex lists working.
//#define GPU_BOUNDING_BOX_CHECK

namespace hdt
{
    // Block size for map type operations (vertex and bounding box calculations). There's no inter-warp
    // reduction in these, so the value isn't very important.
    __host__ __device__ constexpr int cuMapBlockSize() { return 128; }

    // Block size for bounding box reduction. Each warp here is independent - larger blocks just do multiple
    // chunks at once. Should be at least 64 for maximum occupancy.
    __host__ __device__ constexpr int cuReduceBlockSize() { return 64; }

    template<typename T>
    constexpr int collisionBlockSize();

    // Block size for collision checking. Must be a power of 2 for the simple inter-warp reductions to work,
    // and at least 64 for the merge buffer updates.
    template<>
    constexpr int collisionBlockSize<VertexInputArray>() { return 256; }
    template<>
    constexpr int collisionBlockSize<TriangleInputArray>() { return 256; }

    // Maximum number of vertices per patch
    __host__ __device__
    constexpr int vertexListSize() { return 256; }

    // Maximum number of iterations of collision checking with a single vertex list. If there are too many
    // potential collisions to finish in this number of passes, we compute the second vertex list as well.
    __device__
    constexpr int vertexListThresholdFactor() { return 4; }

    __device__ cuVector4::cuVector4()
    {}

    __device__ __forceinline__ cuVector4::cuVector4(float ix, float iy, float iz, float iw)
        : x(ix), y(iy), z(iz), w(iw)
    {}

    __device__ __forceinline__ cuVector4::cuVector4(const cuVector3& v)
        : x(v.x), y(v.y), z(v.z)
    {}

    __device__ __forceinline__ cuVector4::operator cuVector3() const
    {
        return { x, y, z };
    }

    __device__ __forceinline__ cuVector4 cuVector4::operator+(const cuVector4& o) const
    {
        return { x + o.x, y + o.y, z + o.z, w + o.w };
    }

    __device__ __forceinline__ cuVector4 cuVector4::operator-(const cuVector4& o) const
    {
        return { x - o.x, y - o.y, z - o.z, w - o.w };
    }

    __device__ __forceinline__ cuVector4 cuVector4::operator*(const float c) const
    {
        return { x * c, y * c, z * c, w * c };
    }

    __device__ __forceinline__ cuVector4& cuVector4::operator+=(const cuVector4& o)
    {
        *this = *this + o;
        return *this;
    }

    __device__ __forceinline__ cuVector4& cuVector4::operator-=(const cuVector4& o)
    {
        *this = *this - o;
        return *this;
    }

    __device__ __forceinline__ cuVector4& cuVector4::operator *= (const float c)
    {
        *this = *this * c;
        return *this;
    }

    __device__
        cuVector4 crossProduct(const cuVector4& v1, const cuVector4& v2)
    {
        return { v1.y * v2.z - v1.z * v2.y, v1.z * v2.x - v1.x * v2.z, v1.x * v2.y - v1.y * v2.x, 0 };
    }

    __device__
        float dotProduct(const cuVector4& v1, const cuVector4& v2)
    {
        return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
    }

    __device__ float cuVector4::magnitude2() const
    {
        return dotProduct(*this, *this);
    }

    __device__ float cuVector4::magnitude() const
    {
        return sqrt(magnitude2());
    }

    __device__ cuVector4 cuVector4::normalize() const
    {
        return *this * rsqrt(magnitude2());
    }

    __device__ cuVector3::cuVector3(float ix, float iy, float iz)
        : x(ix), y(iy), z(iz)
    {}

    __device__ __forceinline__ cuVector3 perElementMin(const cuVector3& v1, const cuVector4& v2)
    {
        return { min(v1.x, v2.x), min(v1.y, v2.y), min(v1.z, v2.z) };
    }

    __device__ __forceinline__ cuVector3 perElementMax(const cuVector3& v1, const cuVector4& v2)
    {
        return { max(v1.x, v2.x), max(v1.y, v2.y), max(v1.z, v2.z) };
    }

    __device__ __forceinline__ cuVector3 perElementMin(const cuVector3& v1, const cuVector3& v2)
    {
        return { min(v1.x, v2.x), min(v1.y, v2.y), min(v1.z, v2.z) };
    }

    __device__ __forceinline__ cuVector3 perElementMax(const cuVector3& v1, const cuVector3& v2)
    {
        return { max(v1.x, v2.x), max(v1.y, v2.y), max(v1.z, v2.z) };
    }

    __device__ cuAabb3::cuAabb3()
        : aabbMin({ FLT_MAX, FLT_MAX, FLT_MAX }), aabbMax({ -FLT_MAX, -FLT_MAX, -FLT_MAX })
    {}

    __device__ cuAabb3::cuAabb3(const cuVector3& mins, const cuVector3& maxs)
        : aabbMin(mins), aabbMax(maxs)
    {}

    __device__ cuAabb3::cuAabb3(const cuVector4& v)
        : aabbMin(v), aabbMax(v)
    {}

    template<typename... Args>
    __device__ __forceinline__ cuAabb3::cuAabb3(const cuVector4& v, const Args&... args)
        : cuAabb3(args...)
    {
        aabbMin = perElementMin(aabbMin, v);
        aabbMax = perElementMax(aabbMax, v);
    }

    __device__ void cuAabb3::addMargin(const float margin)
    {
        aabbMin.x -= margin;
        aabbMin.y -= margin;
        aabbMin.z -= margin;
        aabbMax.x += margin;
        aabbMax.y += margin;
        aabbMax.z += margin;
    }

    __device__
        bool boundingBoxCollision(const cuAabb3& b1, const cuAabb& b2)
    {
        return !(b1.aabbMin.x > b2.aabbMax.x ||
            b1.aabbMin.y > b2.aabbMax.y ||
            b1.aabbMin.z > b2.aabbMax.z ||
            b1.aabbMax.x < b2.aabbMin.x ||
            b1.aabbMax.y < b2.aabbMin.y ||
            b1.aabbMax.z < b2.aabbMin.z);
    }

    template <unsigned int BlockSize = cuMapBlockSize()>
    __global__ void kernelPerVertexUpdate(
        int n,
        VertexInputArray in,
        BoundingBoxArray out,
        const cuVector4* __restrict__ vertexData)
    {
        int index = blockIdx.x * BlockSize + threadIdx.x;
        int stride = BlockSize * gridDim.x;

        for (int i = index; i < n; i += stride)
        {
            const cuPerVertexInput data = in[i];
            const cuVector4 v = vertexData[data.vertexIndex];
            cuAabb3 aabb(v);
            aabb.addMargin(v.w * data.margin);
            out[i] = aabb;
        }
    }

    template <unsigned int BlockSize = cuMapBlockSize()>
    __global__ void kernelPerTriangleUpdate(
        int n,
        TriangleInputArray in,
        BoundingBoxArray out,
        const cuVector4* __restrict__ vertexData)
    {
        int index = blockIdx.x * BlockSize + threadIdx.x;
        int stride = BlockSize * gridDim.x;

        for (int i = index; i < n; i += stride)
        {
            const cuPerTriangleInput data = in[i];
            const cuVector4 v0 = vertexData[data.vertexIndices.a];
            const cuVector4 v1 = vertexData[data.vertexIndices.b];
            const cuVector4 v2 = vertexData[data.vertexIndices.c];

            float penetration = abs(data.penetration);
            float margin = max((v0.w + v1.w + v2.w) * data.margin / 3, penetration);

            cuAabb3 aabb(v0, v1, v2);
            aabb.addMargin(margin);
            out[i] = aabb;
        }
    }

    template< unsigned int BlockSize = cuReduceBlockSize() >
    __global__ void kernelBoundingBoxReduce(
        int n,
        const std::pair<int, int>* __restrict__ nodeData,
        const BoundingBoxArray boundingBoxes,
        cuAabb* __restrict__ output)
    {
        int tid = threadIdx.x;
        int threadInWarp = tid & 0x1f;
        int warpid = tid >> 5;
        constexpr int nwarps = BlockSize >> 5;
        int stride = gridDim.x * nwarps;

        for (int block = blockIdx.x * nwarps + warpid; block < n; block += stride)
        {
            int firstBox = nodeData[block].first;
            int aabbCount = nodeData[block].second;

            // Load the first block of bounding boxes
            cuAabb3 temp = (threadInWarp < aabbCount) ? boundingBoxes[firstBox + threadInWarp] : cuAabb3();

            // Take union with each successive block
            for (int i = threadInWarp + 32; i < aabbCount; i += 32)
            {
                cuAabb3 box = boundingBoxes[firstBox + i];
                temp.aabbMin = perElementMin(temp.aabbMin, box.aabbMin);
                temp.aabbMax = perElementMax(temp.aabbMax, box.aabbMax);
            }

            // Intra-warp reduce
            for (int j = 16; j > 0; j >>= 1)
            {
                temp.aabbMin.x = min(temp.aabbMin.x, __shfl_down_sync(0xffffffff, temp.aabbMin.x, j));
                temp.aabbMin.y = min(temp.aabbMin.y, __shfl_down_sync(0xffffffff, temp.aabbMin.y, j));
                temp.aabbMin.z = min(temp.aabbMin.z, __shfl_down_sync(0xffffffff, temp.aabbMin.z, j));
                temp.aabbMax.x = max(temp.aabbMax.x, __shfl_down_sync(0xffffffff, temp.aabbMax.x, j));
                temp.aabbMax.y = max(temp.aabbMax.y, __shfl_down_sync(0xffffffff, temp.aabbMax.y, j));
                temp.aabbMax.z = max(temp.aabbMax.z, __shfl_down_sync(0xffffffff, temp.aabbMax.z, j));
            }

            // Store result
            if (threadInWarp == 0)
            {
                output[block].aabbMin = temp.aabbMin;
                output[block].aabbMax = temp.aabbMax;
            }
        }
    }

    template <unsigned int BlockSize = cuMapBlockSize()>
    __global__ void kernelBodyUpdate(
        int n,
        const cuVertex* __restrict__ in,
        cuVector4* __restrict__ out,
        const cuBone* __restrict__ boneData)
    {
        // We work with an entire warp per vertex here
        int index = (blockIdx.x * BlockSize + threadIdx.x) >> 5;
        int stride = (BlockSize * gridDim.x) >> 5;

        // But we do 8 vertices sequentially, so we can write a full 32 values back at the end
        index <<= 3;
        stride <<= 3;

        int tid = threadIdx.x;
        int threadInWarp = tid & 0x1f;
        int threadInHalfWarp = tid & 0x0f;
        int halfWarp = threadInWarp >> 4;
        int eighthWarp = threadInWarp >> 2;
        int element = eighthWarp & 0x03;

        for (int i = index; i < n; i += stride)
        {
            float v;
            float result;
            for (int j = 0; j < 8 && i + j < n; ++j)
            {
                v = boneData[in[i + j].bones[halfWarp]].vals()[threadInHalfWarp] * in[i + j].position.vals()[element] * in[i + j].weights[halfWarp];
                v += boneData[in[i + j].bones[halfWarp + 2]].vals()[threadInHalfWarp] * in[i + j].position.vals()[element] * in[i + j].weights[halfWarp + 2];
                v += __shfl_xor_sync(0xffffffff, v, 4);
                v += __shfl_xor_sync(0xffffffff, v, 8);
                v += __shfl_xor_sync(0xffffffff, v, 16);
                if (eighthWarp == j)
                {
                    result = v;
                }
            }
            if (i + eighthWarp < n)
            {
                // Note we exploit the fact that the output values are contiguous here to (hopefully) do a
                // full 128-byte transaction.
                out[i].vals()[threadInWarp] = result;
            }
        }
    }

    // collidePair does the actual collision between two colliders, always a vertex and some other type. It
    // should modify output if and only if there is a collision.
    template <cuPenetrationType>
    __device__ bool collidePair(
        const cuPerVertexInput& __restrict__ inputA,
        const cuPerVertexInput& __restrict__ inputB,
        const cuVector4* __restrict__ vertexDataA,
        const cuVector4* __restrict__ vertexDataB,
        cuCollisionResult& output)
    {
        const cuVector4 vA = vertexDataA[inputA.vertexIndex];
        const cuVector4 vB = vertexDataB[inputB.vertexIndex];

        float rA = vA.w * inputA.margin;
        float rB = vB.w * inputB.margin;
        float bound2 = (rA + rB) * (rA + rB);
        cuVector4 diff = vA - vB;
        float dist2 = diff.magnitude2();
        float len = sqrt(dist2);
        float dist = len - (rA + rB);
        if (dist2 <= bound2 && (dist < output.depth))
        {
            if (len <= FLT_EPSILON)
            {
                diff = { 1, 0, 0, 0 };
            }
            else
            {
                diff = diff.normalize();
            }
            output.depth = dist;
            output.normOnB = diff;
            output.posA = vA - diff * rA;
            output.posB = vB + diff * rB;
            return true;
        }
        return false;
    }

    template <cuPenetrationType penType>
    __device__ bool collidePair(
        const cuPerVertexInput& __restrict__ inputA,
        const cuPerTriangleInput& __restrict__ inputB,
        const cuVector4* __restrict__ vertexDataA,
        const cuVector4* __restrict__ vertexDataB,
        cuCollisionResult& output)
    {
        cuVector4 s = vertexDataA[inputA.vertexIndex];
        float r = s.w * inputA.margin;
        cuVector4 p0 = vertexDataB[inputB.vertexIndices.a];
        cuVector4 p1 = vertexDataB[inputB.vertexIndices.b];
        cuVector4 p2 = vertexDataB[inputB.vertexIndices.c];
        float margin = (p0.w + p1.w + p2.w) / 3.0;
        float penetration = inputB.penetration * margin;
        margin *= inputB.margin;

        // Compute unit normal and twice area of triangle
        cuVector4 ab = p1 - p0;
        cuVector4 ac = p2 - p0;
        cuVector4 raw_normal = crossProduct(ab, ac);
        float area2 = raw_normal.magnitude2();
        float area = sqrt(area2);

        // Check for degenerate triangles
        if (area < FLT_EPSILON)
        {
            return false;
        }
        cuVector4 normal = raw_normal * (1.0 / area);

        // Compute distance from point to plane and its projection onto the plane
        cuVector4 ap = s - p0;
        float distance = dotProduct(ap, normal);

        float radiusWithMargin = r + margin;
        if (penType == eNone)
        {
            // Two-sided check: make sure distance is positive and normal is in the correct direction
            if (distance < 0)
            {
                distance = -distance;
                normal *= -1.0;
            }
        }
        else if (distance < penetration)
        {
            // One-sided check: make sure sphere center isn't too far on the wrong side of the triangle
            return false;
        }

        // Don't bother to do any more if there's no collision or we already have a deeper one
        float depth = distance - radiusWithMargin;
        if (depth >= -FLT_EPSILON || depth >= output.depth)
        {
            return false;
        }

        // Compute triple products and check the projection lies in the triangle
        cuVector4 bp = s - p1;
        cuVector4 cp = s - p2;
        ac = crossProduct(ap, bp);
        ab = crossProduct(cp, ap);
        float areaC = dotProduct(ac, raw_normal);
        float areaB = dotProduct(ab, raw_normal);
        float areaA = area2 - areaB - areaC;
        if (areaA < 0 || areaB < 0 || areaC < 0)
        {
            return false;
        }

        output.normOnB = normal;
        output.posA = s - normal * r;
        output.posB = s - normal * (distance - margin);
        output.depth = depth;
        return true;
    }

    template<int BlockSize>
    __device__ int kernelComputeVertexList(
        int start,
        int n,
        int tid,
        const BoundingBoxArray boundingBoxes,
        const cuAabb& boundingBox,
        int* intShared,
        int* vertexList
    )
    {
        int* partialSums = intShared + 32;
        int threadInWarp = tid & 0x1f;
        int warpid = tid >> 5;
        constexpr int nwarps = BlockSize >> 5;

        // Set up vertex list for shape
        int nCeil = (((n - 1) / BlockSize) + 1) * BlockSize;
        int blockStart = 0;
        for (int i = tid; blockStart < vertexListSize() && i < nCeil; i += BlockSize)
        {
            int vertex = i + start;
            bool collision = i < n && boundingBoxCollision(boundingBoxes[vertex], boundingBox);

            // Count the number of collisions in this warp and store in shared memory
            auto mask = __ballot_sync(0xffffffff, collision);
            if (threadInWarp == 0)
            {
                intShared[warpid] = __popc(mask);
            }
            __syncthreads();

            // Compute partial sum counts for warps
            if (warpid == 0)
            {
                int a = intShared[threadInWarp];
                for (int j = 1; j < nwarps; j <<= 1)
                {
                    int b = __shfl_up_sync(0xffffffff, a, j);
                    if (threadInWarp >= j)
                    {
                        a += b;
                    }
                }
                partialSums[threadInWarp] = blockStart + a;
            }

            __syncthreads();

            // Now we can calculate where to put the index, if it's a potential collision
            if (collision)
            {
                int warpStart = (warpid > 0) ? partialSums[warpid - 1] : blockStart;
                unsigned int lanemask = (1UL << threadInWarp) - 1;
                int index = warpStart + __popc(mask & lanemask);
                if (index < vertexListSize())
                {
                    vertexList[index] = vertex;
                }
            }

            blockStart = partialSums[nwarps - 1];
        }

        // Update number of colliders in A and the total number of pairs
        return min(blockStart, vertexListSize());
    }

    template<int BlockSize>
    __device__ int kernelPopulateVertexList(
        int start,
        int n,
        int tid,
        int* vertexList
    )
    {
        int size = min(n, vertexListSize());
        if (tid < size)
        {
            vertexList[tid] = start + tid;
        }
        return size;
    }

    template<typename T>
    __device__ constexpr int BoneCount();
    template<>
    __device__ constexpr int BoneCount<cuPerVertexInput>() { return 4; }
    template<>
    __device__ constexpr int BoneCount<cuPerTriangleInput>() { return 12; }

    __device__ uint32_t getBone(const cuVertex* vertexSetup, const cuPerVertexInput& collider, int i)
    {
        return vertexSetup[collider.vertexIndex].bones[i];
    }

    __device__ uint32_t getBone(const cuVertex* vertexSetup, const cuPerTriangleInput& collider, int i)
    {
        int index = (i < 4) ? collider.vertexIndices.a
            : (i < 8) ? collider.vertexIndices.b
            : collider.vertexIndices.c;

        return vertexSetup[index].bones[i & 3];
    }

    __device__ float getBoneWeight(const cuVertex* vertexSetup, const cuPerVertexInput& collider, int i)
    {
        return vertexSetup[collider.vertexIndex].weights[i];
    }

    __device__ float getBoneWeight(const cuVertex* vertexSetup, const cuPerTriangleInput& collider, int i)
    {
        int index = (i < 4) ? collider.vertexIndices.a
            : (i < 8) ? collider.vertexIndices.b
            : collider.vertexIndices.c;

        return vertexSetup[index].weights[i & 3];
    }

    // kernelCollision does the supporting work for threading the collision checks and making sure that only
    // the deepest result is kept.
    template <cuPenetrationType penType = eNone, typename T, int BlockSize = collisionBlockSize<T>()>
    __global__ void __launch_bounds__(collisionBlockSize<T>(), 1024 / collisionBlockSize<T>()) kernelCollision(
        int n,
        bool swap,
        const cuCollisionSetup* __restrict__ setup,
        const VertexInputArray inA,
        const T inB,
        const BoundingBoxArray boundingBoxesA,
        const BoundingBoxArray boundingBoxesB,
        const cuVertex* __restrict__ vertexSetupA,
        const cuVertex* __restrict__ vertexSetupB,
        const cuVector4* __restrict__ vertexDataA,
        const cuVector4* __restrict__ vertexDataB,
        const float* __restrict__ boneWeightsA,
        const float* __restrict__ boneWeightsB,
        const int* __restrict__ boneMapA,
        const int* __restrict__ boneMapB,
        cuCollisionMerge* mergeBuffer,
        int mergeX,
        int mergeDynX,
        int mergeY)
    {
        static_assert(vertexListSize() <= BlockSize, "Vertex list must be smaller than block size");

        __shared__ float floatShared[64 + 2 * vertexListSize()];
        int* intShared = reinterpret_cast<int*>(floatShared);

        int tid = threadIdx.x;
        int threadInWarp = tid & 0x1f;
        int warpid = tid >> 5;
        constexpr int nwarps = BlockSize >> 5;
        
        for (int block = blockIdx.x; block < n; block += gridDim.x)
        {
            int nA = setup[block].sizeA;
            int nB = setup[block].sizeB; 
            int offsetA = setup[block].offsetA;
            int offsetB = setup[block].offsetB;

            // Depth should always be negative for collisions. We'll use positive values to signify no
            // collision, and later for mutual exclusion.
            cuCollisionResult temp;
            temp.depth = 1;

            int* vertexListA = intShared + 64;
            int* vertexListB = vertexListA + vertexListSize();

            // Calculate or populate vertex lists, if the number of possible pairs is large. Start with the
            // larger one, and only do the second if the number of pairs is still too high.
            bool order = nA > nB;
            if (order)
            {
                if (nA * nB > BlockSize * vertexListThresholdFactor())
                {
                    nA = kernelComputeVertexList<BlockSize>(
                        offsetA,
                        nA,
                        tid,
                        boundingBoxesA,
                        setup[block].boundingBoxB,
                        intShared,
                        vertexListA);
                }
                else
                {
                    nA = kernelPopulateVertexList<BlockSize>(offsetA, nA, tid, vertexListA);
                }
            }

            if (nA * nB > BlockSize * vertexListThresholdFactor())
            {
                nB = kernelComputeVertexList<BlockSize>(
                    offsetB,
                    nB,
                    tid,
                    boundingBoxesB,
                    setup[block].boundingBoxA,
                    intShared,
                    vertexListB);
            }
            else
            {
                nB = kernelPopulateVertexList<BlockSize>(offsetB, nB, tid, vertexListB);
            }

            if (!order)
            {
                if (nA * nB > BlockSize * vertexListThresholdFactor())
                {
                    nA = kernelComputeVertexList<BlockSize>(
                        offsetA,
                        nA,
                        tid,
                        boundingBoxesA,
                        setup[block].boundingBoxB,
                        intShared,
                        vertexListA);
                }
                else
                {
                    nA = kernelPopulateVertexList<BlockSize>(offsetA, nA, tid, vertexListA);
                }
            }

            // kernelComputeVertexList doesn't do a final synchronize, because it's OK to run it
            // sequentially for both lists without synchronizing between them. So we need to synchronize
            // now to make sure the vertex lists are fully visible.
            __syncthreads();

            int nPairs = nA * nB;

            for (int i = tid; i < nPairs; i += BlockSize)
            {
                int iA = vertexListA[i % nA];
                int iB = vertexListB[i / nA];

                // Skip pairs until we find one with a bounding box collision. This should increase the
                // number of full checks done in parallel, and reduce divergence overall. Note we only do
                // this at all if there are more pairs than threads - if there's only enough work for a
                // single iteration (very common), there's no benefit to trying to reduce it.
#ifdef GPU_BOUNDING_BOX_CHECK
                if (nPairs > BlockSize)
                {
                    while (i < nPairs && !boundingBoxCollision(boundingBoxesA[iA], boundingBoxesB[iB]))
                    {
                        i += BlockSize;
                        if (i < nPairs)
                        {
                            iA = vertexListA[i % nA];
                            iB = vertexListB[i / nA];
                        }
                    }
                }
#endif

                if (i < nPairs && collidePair<penType>(inA[iA], inB[iB], vertexDataA, vertexDataB, temp))
                {
                    temp.colliderA = iA;
                    temp.colliderB = iB;
                }
            }

            // Find minimum depth in this warp and store in shared memory
            float d = temp.depth;
            for (int j = 16; j > 0; j >>= 1)
            {
                d = min(d, __shfl_down_sync(0xffffffff, d, j));
            }
            if (threadInWarp == 0)
            {
                floatShared[warpid] = d;
            }
            __syncthreads();

            // Find minimum across warps
            if (warpid == 0)
            {
                d = floatShared[threadInWarp];
                for (int j = nwarps >> 1; j > 0; j >>= 1)
                {
                    d = min(d, __shfl_down_sync(0xffffffff, d, j));
                }
                if (threadInWarp == 0)
                {
                    floatShared[0] = d;
                }
            }
            __syncthreads();

            if (floatShared[0] > -FLT_EPSILON)
            {
                return;
            }

            // If the depth of this thread is equal to the minimum, try to set the result. Do an atomic
            // exchange with the first value to ensure that only one thread gets to do this in case of ties.
            cuCollisionResult* result = reinterpret_cast<cuCollisionResult*>(floatShared + 32);
            if (floatShared[0] == temp.depth && atomicExch(floatShared, 2) == temp.depth)
            {
                *result = temp;
            }

            __syncthreads();

            // Update cumulative values in the merge buffer. Use the first two warps, each processing eight
            // or twenty-four entries, depending on the type of collision.
            int indexA = threadIdx.x >> 4;
            int indexB = threadIdx.x & 0x0f;
            if (indexA < BoneCount<cuPerVertexInput>() && indexB < BoneCount<T::type>())
            {
                uint32_t boneA = getBone(vertexSetupA, inA[result->colliderA], indexA);
                uint32_t boneB = getBone(vertexSetupB, inB[result->colliderB], indexB);

                float weightA = getBoneWeight(vertexSetupA, inA[result->colliderA], indexA);
                float weightB = getBoneWeight(vertexSetupB, inB[result->colliderB], indexB);

                if (weightA <= boneWeightsA[boneA] || weightB <= boneWeightsB[boneB])
                {
                    return;
                }

                // FIXME: Get this from collider data, if it even does anything useful
                float flexible = 1.0;

                float w = flexible * result->depth;
                float w2 = w * w;

                int i = swap ? boneB : boneA;
                int i_map = swap ? boneMapB[boneB] : boneMapA[boneA];
                int j = swap ? boneA : boneB;
                int j_map = swap ? boneMapA[boneA] : boneMapB[boneB];

                cuCollisionMerge* c;

                if (i_map == -1 && j_map != -1)
                {
                    c = mergeBuffer + mergeDynX * mergeY + mergeX * j_map + i;
                }
                else if (i_map != -1)
                {
                    c = mergeBuffer + i_map * mergeY + j;
                }
                else
                {
                    return;
                }

                atomicAdd(&c->weight, w2);

                if (swap)
                {
                    atomicAdd(&c->normal.x, -result->normOnB.x * w * w2);
                    atomicAdd(&c->normal.y, -result->normOnB.y * w * w2);
                    atomicAdd(&c->normal.z, -result->normOnB.z * w * w2);
                    atomicAdd(&c->normal.w, -result->normOnB.w * w * w2);
                    atomicAdd(&c->posA.x, result->posB.x * w2);
                    atomicAdd(&c->posA.y, result->posB.y * w2);
                    atomicAdd(&c->posA.z, result->posB.z * w2);
                    atomicAdd(&c->posA.w, result->posB.w * w2);
                    atomicAdd(&c->posB.x, result->posA.x * w2);
                    atomicAdd(&c->posB.y, result->posA.y * w2);
                    atomicAdd(&c->posB.z, result->posA.z * w2);
                    atomicAdd(&c->posB.w, result->posA.w * w2);
                }
                else
                {
                    atomicAdd(&c->normal.x, result->normOnB.x * w * w2);
                    atomicAdd(&c->normal.y, result->normOnB.y * w * w2);
                    atomicAdd(&c->normal.z, result->normOnB.z * w * w2);
                    atomicAdd(&c->normal.w, result->normOnB.w * w * w2);
                    atomicAdd(&c->posA.x, result->posA.x * w2);
                    atomicAdd(&c->posA.y, result->posA.y * w2);
                    atomicAdd(&c->posA.z, result->posA.z * w2);
                    atomicAdd(&c->posA.w, result->posA.w * w2);
                    atomicAdd(&c->posB.x, result->posB.x * w2);
                    atomicAdd(&c->posB.y, result->posB.y * w2);
                    atomicAdd(&c->posB.z, result->posB.z * w2);
                    atomicAdd(&c->posB.w, result->posB.w * w2);
                }
            }
        }
    }

    __global__ void fullInternalUpdate(
        int nVertices,
        const cuVertex* __restrict__ verticesIn,
        cuVector4* vertexData,
        const cuBone* __restrict__ boneData,
        int nVertexColliders,
        VertexInputArray perVertexIn,
        BoundingBoxArray perVertexOut,
        int nVertexNodes,
        const std::pair<int, int>* __restrict__ vertexNodeData,
        cuAabb* vertexNodeOutput,
        int nTriangleColliders,
        TriangleInputArray perTriangleIn,
        BoundingBoxArray perTriangleOut,
        int nTriangleNodes,
        const std::pair<int, int>* __restrict__ triangleNodeData,
        cuAabb* triangleNodeOutput )
    {
        if (threadIdx.x == 0)
        {
            // Each warp of 32 threads processes 8 vertices sequentially, so we need 4 threads per vertex
            int nBodyBlocks = (nVertices * 4 - 1) / cuMapBlockSize() + 1;
            kernelBodyUpdate <<<nBodyBlocks, cuMapBlockSize()>>> (nVertices, verticesIn, vertexData, boneData);

            constexpr int warpsPerBlock = cuReduceBlockSize() >> 5;

            if (nVertexColliders > 0)
            {
                int nVertexBlocks = (nVertexColliders - 1) / cuMapBlockSize() + 1;
                kernelPerVertexUpdate <<<nVertexBlocks, cuMapBlockSize(), 0>>> (nVertexColliders, perVertexIn, perVertexOut, vertexData);
                int nReduceBlocks = ((nVertexNodes - 1) / warpsPerBlock) + 1;
                kernelBoundingBoxReduce <<<nReduceBlocks, cuReduceBlockSize(), 0>>> (nVertexNodes, vertexNodeData, perVertexOut, vertexNodeOutput);
            }
            if (nTriangleColliders > 0)
            {
                int nTriangleBlocks = (nTriangleColliders - 1) / cuMapBlockSize() + 1;
                kernelPerTriangleUpdate <<<nTriangleBlocks, cuMapBlockSize(), 0>>> (nTriangleColliders, perTriangleIn, perTriangleOut, vertexData);
                int nReduceBlocks = ((nTriangleNodes - 1) / warpsPerBlock) + 1;
                kernelBoundingBoxReduce <<<nReduceBlocks, cuReduceBlockSize(), 0>>> (nTriangleNodes, triangleNodeData, perTriangleOut, triangleNodeOutput);
            }
        }
    }

    cuResult cuCreateStream(void** ptr)
    {
        *ptr = new hipStream_t;
        return hipStreamCreate(reinterpret_cast<hipStream_t*>(*ptr));
    }

    void cuDestroyStream(void* ptr)
    {
        hipStreamDestroy(*reinterpret_cast<hipStream_t*>(ptr));
        delete reinterpret_cast<hipStream_t*>(ptr);
    }

    cuResult cuGetDeviceBuffer(void** buf, int size)
    {
        return hipMalloc(buf, size);
    }

    cuResult cuGetHostBuffer(void** buf, int size)
    {
        return hipHostMalloc(buf, size);
    }

    void cuFreeDevice(void* buf)
    {
        hipFree(buf);
    }

    void cuFreeHost(void* buf)
    {
        hipHostFree(buf);
    }

    cuResult cuCopyToDevice(void* dst, void* src, size_t n, void* stream)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        return hipMemcpyAsync(dst, src, n, hipMemcpyHostToDevice, *s);
    }

    cuResult cuCopyToHost(void* dst, void* src, size_t n, void* stream)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        return hipMemcpyAsync(dst, src, n, hipMemcpyDeviceToHost, *s);
    }

    cuResult cuMemset(void* buf, int value, size_t n, void* stream)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        return hipMemsetAsync(buf, value, n, *s);
    }

    template<cuPenetrationType penType, typename T>
    cuResult cuRunCollision(
        void* stream,
        int n,
        bool swap,
        cuCollisionSetup* setup,
        VertexInputArray inA,
        T inB,
        BoundingBoxArray boundingBoxesA,
        BoundingBoxArray boundingBoxesB,
        cuVertex* vertexSetupA,
        cuVertex* vertexSetupB,
        cuVector4* vertexDataA,
        cuVector4* vertexDataB,
        float* boneWeightsA,
        float* boneWeightsB,
        int* boneMapA,
        int* boneMapB,
        cuCollisionMerge* mergeBuffer,
        int mergeX,
        int mergeDynX,
        int mergeY)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);

        kernelCollision<penType> <<<n, collisionBlockSize<T>(), 0, *s >>> (
            n, swap, setup, inA, inB, boundingBoxesA, boundingBoxesB, vertexSetupA, vertexSetupB, vertexDataA, vertexDataB, boneWeightsA, boneWeightsB, boneMapA, boneMapB, mergeBuffer, mergeX, mergeDynX, mergeY);
        return cuResult();
    }

    cuResult cuInternalUpdate(
        void* stream,
        int nVertices,
        const cuVertex* verticesIn,
        cuVector4* vertexData,
        const cuBone* boneData,
        int nVertexColliders,
        VertexInputArray perVertexIn,
        BoundingBoxArray perVertexOut,
        int nVertexNodes,
        const std::pair<int, int>* vertexNodeData,
        cuAabb* vertexNodeOutput,
        int nTriangleColliders,
        TriangleInputArray perTriangleIn,
        BoundingBoxArray perTriangleOut,
        int nTriangleNodes,
        const std::pair<int, int>* triangleNodeData,
        cuAabb* triangleNodeOutput)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);

        fullInternalUpdate <<<1, 1, 0, *s >>> (
            nVertices,
            verticesIn,
            vertexData,
            boneData,
            nVertexColliders,
            perVertexIn,
            perVertexOut,
            nVertexNodes,
            vertexNodeData,
            vertexNodeOutput,
            nTriangleColliders,
            perTriangleIn,
            perTriangleOut,
            nTriangleNodes,
            triangleNodeData,
            triangleNodeOutput);
        return cuResult();
    }

    cuResult cuSynchronize(void* stream)
    {
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);

        if (s)
        {
            return hipStreamSynchronize(*s);
        }
        else
        {
            return hipDeviceSynchronize();
        }
    }

    cuResult cuCreateEvent(void** ptr)
    {
        *ptr = new hipEvent_t;
        return hipEventCreate(reinterpret_cast<hipEvent_t*>(*ptr));
    }

    void cuDestroyEvent(void* ptr)
    {
        hipEventDestroy(*reinterpret_cast<hipEvent_t*>(ptr));
        delete reinterpret_cast<hipEvent_t*>(ptr);
    }

    void cuRecordEvent(void* ptr, void* stream)
    {
        hipEvent_t* e = reinterpret_cast<hipEvent_t*>(ptr);
        hipStream_t* s = reinterpret_cast<hipStream_t*>(stream);
        hipEventRecord(*e, *s);
    }

    void cuWaitEvent(void* ptr)
    {
        hipEvent_t* e = reinterpret_cast<hipEvent_t*>(ptr);
        hipEventSynchronize(*e);
    }

    void cuInitialize()
    {
//        hipSetDeviceFlags(hipDeviceScheduleYield);
    }

    int cuDeviceCount()
    {
        int count;
        hipGetDeviceCount(&count);
        return count;
    }

    void cuSetDevice(int id)
    {
        hipSetDevice(id);
    }

    int cuGetDevice()
    {
        int id;
        hipGetDevice(&id);
        return id;
    }

    template cuResult cuRunCollision<eNone, VertexInputArray>(
        void*, int, bool, cuCollisionSetup*, VertexInputArray, VertexInputArray, BoundingBoxArray, BoundingBoxArray,
        cuVertex*, cuVertex*, cuVector4*, cuVector4*,
        float*, float*, int*, int*, cuCollisionMerge*, int, int, int);
    template cuResult cuRunCollision<eNone, TriangleInputArray>(
        void*, int, bool, cuCollisionSetup*, VertexInputArray, TriangleInputArray, BoundingBoxArray, BoundingBoxArray,
        cuVertex*, cuVertex*, cuVector4*, cuVector4*,
        float*, float*, int*, int*, cuCollisionMerge*, int, int, int);
    template cuResult cuRunCollision<eInternal, TriangleInputArray>(
        void*, int, bool, cuCollisionSetup*, VertexInputArray, TriangleInputArray, BoundingBoxArray, BoundingBoxArray,
        cuVertex*, cuVertex*, cuVector4*, cuVector4*,
        float*, float*, int*, int*, cuCollisionMerge*, int, int, int);
}
